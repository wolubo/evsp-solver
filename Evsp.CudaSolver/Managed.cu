#include "Managed.h"

#include "hip/hip_runtime.h"
#include "CudaCheck.h"





void * Managed::operator new(size_t len) {
	void *ptr;
	CUDA_CHECK(hipMallocManaged(&ptr, len));
	CUDA_CHECK(hipDeviceSynchronize());
	return ptr;
}

void Managed::operator delete(void *ptr) {
	CUDA_CHECK(hipDeviceSynchronize());
	CUDA_CHECK(wbCudaFree(__FILE__, __LINE__, ptr));
}

void* Managed::operator new[](size_t len)
{
	void *ptr;
	CUDA_CHECK(hipMallocManaged(&ptr, len));
	CUDA_CHECK(hipDeviceSynchronize());
	return ptr;
}

void Managed::operator delete[](void *ptr)
{
	CUDA_CHECK(hipDeviceSynchronize());
	CUDA_CHECK(wbCudaFree(__FILE__, __LINE__, ptr));
}

