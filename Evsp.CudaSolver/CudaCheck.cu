#include "CudaCheck.h"

#include <stdio.h>
#include "hip/hip_runtime.h"


hipError_t wbCudaMalloc(const char *file, int line, void **devPtr, size_t size)
{
	hipError_t code = hipMalloc(devPtr, size);

#ifdef WB_MEMCHECK
	printf("hipMalloc: %p, %u bytes allocated (%s %i)\n", *devPtr, size, file, line);
#endif

	return code;
}


hipError_t wbCudaFree(const char *file, int line, void *devPtr)
{
	hipError_t code = hipFree(devPtr);

#ifdef WB_MEMCHECK
	printf("hipFree: %p (%s %d)\n", devPtr, file, line);
#endif

	return code;
}



#ifdef _DEBUG

void printMemStat(const char *file, int line)
{
	size_t free, total;
	float ffree, ftotal;
	CUDA_CHECK(hipMemGetInfo(&free, &total));
	ffree = free / (1024.0f * 1024.0f);
	ftotal = total / (1024.0f * 1024.0f);
	printf("Memstat: %.0f mb of %.0f mb free (%s %d)\n", ffree, ftotal, file, line);
}
#endif

__host__ void gpuAssert(hipError_t code, const char *file, int line)
{
	if (code != hipSuccess) {
		printf("GPUassert: %s (Code: %d) %s %d\n", hipGetErrorString(code), code, file, line);
		hipDeviceReset();
		exit(code);
	}
}

__device__ void gpuAssertDev(hipError_t code, const char *file, int line)
{
	if (code != hipSuccess) {
		printf("GPUassert: %i %s %d\n", code, file, line);
	}
}


